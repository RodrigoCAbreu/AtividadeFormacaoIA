
#include <hip/hip_runtime.h>
#include <stdio.h>

// Número de elementos em cada vetor
#define N 2048 * 2048

__global__ void my_kernel(float scalar, float * x, float * y)
{
    // Determina a identificação de thread global exclusiva, por isso sabemos qual elemento processar
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Certifique-se de que ainda temos threads disponíveis!
    if ( tid < N ) 
        y[tid] = scalar * x[tid] + y[tid];
}

int main()
{
    float *x, *y;

    // O número total de bytes por vetor
    int size = N * sizeof (float); 

    hipError_t ierrAsync;
    hipError_t ierrSync;

    // Aloca memória
    hipMallocManaged(&x, size);
    hipMallocManaged(&y, size);

    // Inicializa a memória
    for( int i = 0; i < N; ++i )
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int threads_per_block = 256;
    int number_of_blocks = (N / threads_per_block) + 1;

    my_kernel <<< number_of_blocks, threads_per_block >>> ( 2.0f, x, y );

    ierrSync = hipGetLastError();

    // Aguarde até que a GPU termine
    ierrAsync = hipDeviceSynchronize(); 

    // Verifica status de execução
    if (ierrSync != hipSuccess) { printf("Sync error: %s\n", hipGetErrorString(ierrSync)); }
    if (ierrAsync != hipSuccess) { printf("Async error: %s\n", hipGetErrorString(ierrAsync)); }

    // Imprime o erro máximo
    float maxError = 0;
    for( int i = 0; i < N; ++i )
        if (abs(4-y[i]) > maxError) { maxError = abs(4-y[i]); }
    printf("Max Error: %.5f", maxError);

    // Libera a memória alocada
    hipFree( x ); hipFree( y );
}