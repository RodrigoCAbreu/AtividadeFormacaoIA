#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "sm_20_atomic_functions.h"

#include <iostream>
using namespace std;

__device__ int dSum = 0;

__global__ void sum(int* d)
{
	int tid = threadIdx.x;
	//dSum += d[tid];

	atomicAdd(&dSum, d[tid]);
}

int main()
{
	const int count = 256;
	const int size = count * sizeof(int);

	int h[count];
	for (int i = 0; i < count; ++i)
		h[i] = i + 1;

	int* d;
	hipMalloc(&d, size);
	hipMemcpy(d, h, size, hipMemcpyHostToDevice);

	// Define 2 eventos CUDA
	hipEvent_t start, end;

	// Cria os eventos
	hipEventCreate(&start);
	hipEventCreate(&end);

	// Registra o primeiro evento
	hipEventRecord(start);

	sum << <1, count >> >(d);

	// Registra o segundo evento
	hipEventRecord(end);

	// Sincroniza o evento
	hipEventSynchronize(end);

	// Calcula o tempo usado no processamento
	float elapsed;
	hipEventElapsedTime(&elapsed, start, end);

	int hSum;
	hipMemcpyFromSymbol(&hSum, HIP_SYMBOL(dSum), sizeof(int));
	cout << "A soma dos valores de 1 a  " << count
		<< " igual a " << hSum << " e foi processada em " << elapsed << " msec" << endl;
	getchar();

	hipFree(d);

	return 0;
}