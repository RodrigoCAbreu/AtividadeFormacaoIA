
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

void addArrays(int* a, int* b, int* c, int count)
{
	for (int i = 0; i < count; ++i)
		c[i] = a[i] + b[i];
}


void main()
{
	// Constante
	const int count = 5;

	// Arrays
	int a[] = { 1, 2, 3, 4, 5 };
	int b[] = { 100, 200, 300, 400, 500 };

	// Arrays para o resultado
	int c[count];

	// Somar os arrays
	addArrays(a, b, c, count);

	// Imprime os itens do array c
	for (int i = 0; i < count; ++i)
		printf("%d ", c[i]);

	getchar();

}

