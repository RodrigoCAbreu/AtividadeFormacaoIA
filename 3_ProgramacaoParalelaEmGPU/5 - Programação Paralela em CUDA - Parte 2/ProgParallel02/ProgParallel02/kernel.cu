#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void addArraysGPU(int* a, int* b, int* c)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void main()
{
	// Constante
	const int count = 5;
	const int size = count * sizeof(int);

	// Arrays - Mem�ria RAM
	int ha[] = { 1, 2, 3, 4, 5 };
	int hb[] = { 100, 200, 300, 400, 500 };

	// Array para gravar o resultado - Mem�ria RAM
	int hc[count];

	// Vari�veis para execu��o na GPU
	int *da, *db, *dc;

	// Aloca��o de mem�ria na GPU
	hipMalloc(&da, size);
	hipMalloc(&db, size);
	hipMalloc(&dc, size);

	// C�pia das vari�veis a e b da Mem�ria RAM para a Mem�ria na GPU
	hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
	hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

	// Definindo um bloco de threads
	addArraysGPU <<<1, count >>>(da, db, dc);

	// C�pia do resultado da Mem�ria da GPU de volta para a Mem�ria da CPU
	hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);

	// Imprime os resultados
	printf("%d %d %d %d %d",
		hc[0],
		hc[1],
		hc[2],
		hc[3],
		hc[4]);

	// Libera as �reas de mem�ria
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	// Para visualizar o resultado na tela at� pressionar uma tecla
	getchar();

}