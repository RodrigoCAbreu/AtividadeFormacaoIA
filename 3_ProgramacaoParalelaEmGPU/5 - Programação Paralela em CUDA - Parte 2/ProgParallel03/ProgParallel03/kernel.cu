
#include "hip/hip_runtime.h"


#include <iostream>
using namespace std;

int main()
{
	int count;
	hipGetDeviceCount(&count);

	hipDeviceProp_t prop;

	for (int i = 0; i < count; ++i)
	{
		hipGetDeviceProperties(&prop, i);

		cout << "Device " << i << ": " << prop.name << endl;
		cout << "Compute Capability: " << prop.major << "." << prop.minor << endl;
		cout << "Max Grid Dimensions: (" << 
			prop.maxGridSize[0] << " x " <<
			prop.maxGridSize[1] << " x " <<
			prop.maxGridSize[2] << " ) " << endl;
		cout << "Max Block Dimensions: (" << 
			prop.maxThreadsDim[0] << " x " <<
			prop.maxThreadsDim[1] << " x " <<
			prop.maxThreadsDim[2] << " ) " << endl;
		cout << "Warp Size: " << prop.warpSize << endl;


	}

	getchar();

	return 0;
}


